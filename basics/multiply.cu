// https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <random>

__global__
void multiply(int n, float *x, float *y, float *z) {
    int index = blockIdx.x * blockDim.x +  threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        z[i] = x[i] * y[i];
    }
}

int main() {
    int N = 1 << 20;
    float *x, *y, *z;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));
    hipMallocManaged(&z, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = (float)(random() % N) / (float) N;
        y[i] = (float)(random() % N) / (float) N;
    }

    multiply<<<1, 1>>>(N, x, y, z);
    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(z[i] - (x[i] * y[i])));
    }
    std::cout << "Max error " << maxError << std::endl;
    hipFree(x);
    hipFree(y);
    hipFree(z);
}